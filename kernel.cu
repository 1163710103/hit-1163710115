# pragma warning (disable:4819)

#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>

#define ARRAYSIZE 5

#define checkCudaErrors( a ) do { \
    if (hipSuccess != (a)) { \
    fprintf(stderr, "Cuda runtime error in line %d of file %s \
    : %s \n", __LINE__, __FILE__, hipGetErrorString(hipGetLastError()) ); \
    exit(EXIT_FAILURE); \
    } \
    } while(0);


void printDeviceProp(hipDeviceProp_t &prop);
void printVector(const int vector[]);

void inquireGPUInfo() {

	int count;

	hipGetDeviceCount(&count); 
	if (count == 0) {
		printf("There is no device.\n");
		return;
	} else {
		printf("Device count is %d.\n\n", count);
	}

	// find the device
	int i;
	for (i = 0; i < count; ++i) {
		hipDeviceProp_t prop;
		if (hipGetDeviceProperties(&prop, i) == hipSuccess) {
				printDeviceProp(prop);
		}
	}
	printf("\n");
}

void printDeviceProp(hipDeviceProp_t &prop)
{
	printf("Device name :\t %s.\n", prop.name);
	printf("Major compute capability: \t %d.\n", prop.major);
	printf("Total global memory: \t %lld bytes.\n", prop.totalGlobalMem);
	printf("Max threads per block: \t %d.\n", prop.maxThreadsPerBlock);
	printf("Total const memory: \t %lld bytes.\n", prop.totalConstMem);
	printf("Shared memory per block: \t %lld bytes.\n", prop.sharedMemPerBlock);
	printf("Registers per block: \t %d.\n", prop.regsPerBlock);
	printf("Max threads per multiprocessors: \t %d.\n", prop.maxThreadsPerMultiProcessor);
	printf("Multiprocessors count: \t %d.\n", prop.multiProcessorCount);
}

__global__ void addKernel(int *c, const int *a, const int *b) {
	int i = threadIdx.x;
	c[i] = a[i] + b[i];
}

void printVector(const int vector[]) {
	int i;
	for (i = 0; i < ARRAYSIZE; i++) {
		if (i != ARRAYSIZE) {
			printf("%d,  ", vector[i]);
		}
		else {
			printf("%d ", vector[i]);
		}
	}
	printf("\n");
}

int main() {
//	inquireGPUInfo();

	const int a[ARRAYSIZE] = { 1, 2, 3, 4, 5 };
	const int b[ARRAYSIZE] = { 10, 20, 30, 40, 50 };
	int c[ARRAYSIZE] = { 0 };

	int *dev_a, *dev_b, *dev_c;
	checkCudaErrors(hipMalloc((void**)&dev_a, ARRAYSIZE * sizeof(int)));
	checkCudaErrors(hipMalloc((void**)&dev_b, ARRAYSIZE * sizeof(int)));
	checkCudaErrors(hipMalloc((void**)&dev_c, ARRAYSIZE * sizeof(int)));

	checkCudaErrors(hipMemcpy(dev_a, a, ARRAYSIZE * sizeof(int)
		, hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_b, b, ARRAYSIZE * sizeof(int)
		, hipMemcpyHostToDevice));

	addKernel <<< 1, ARRAYSIZE >>> (dev_c, dev_a, dev_b);

	checkCudaErrors(hipMemcpy(c, dev_c, ARRAYSIZE * sizeof(int)
		, hipMemcpyDeviceToHost));

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);

	printf("Array 1: \t");
	printVector(a);
	printf("Array 2: \t");
	printVector(b);
	printf("Arrays sum: \t");
	printVector(c);

	getchar();
	return 0;
}